
#include "hip/hip_runtime.h"
#include ""

#define WIN32_LEAN_AND_MEAN
#define VC_EXTRALEAN
#include <Windows.h>

#include <stdio.h>
#include <fstream>
#include <iostream>
#include <sstream>
#include <time.h>

#include <opencv2/highgui/highgui.hpp>


#include "ImageBatch.h"

#include "GPU/Converter.cuh"
#include "GPU/GpuBuffer.cuh"
#include "GPU/ImageConvolution.cuh"
#include "GPU/VarianceCenterizer.cuh"
#include "GPU/Sharpener.cuh"
#include "GPU/MaxPooling.cuh"

#include "Types.h"
#include "Utils/FoldsFactory.h"


//#define MEASURE_SEPARATE


void doUchar(
	std::shared_ptr<cnn::ImageBatch<uchar>>& pImages, 
	std::shared_ptr<cnn::ImageBatch<uchar>>& pKernels);


void doFloat(
	std::shared_ptr<cnn::ImageBatch<uchar>>& b, 
	std::shared_ptr<cnn::ImageBatch<uchar>>& filtersUchar);

int main()
{
	srand((uint32)time(0));
	
	__int64 freq;
	QueryPerformanceFrequency(reinterpret_cast<LARGE_INTEGER*>(&freq));
	double spc = 1.0 / freq;

	/*
	std::string names[] = {
		"9336923", "9338535", "anpage", "asamma", "asewil",
		"astefa", "drbost", "ekavaz", "elduns", "kaknig", 
		"klclar", "ksunth", "lfso", "mbutle", "phughe", 
		"sbains", "slbirc", "vstros", "yfhsie"};
	*/
	std::string names[] = {"slbirc"};
	size_t nsize = ARRAYSIZE(names);

	std::vector<std::string> files;
	for(size_t a=0UL; a<1UL; ++a){
		for(size_t n=0UL; n<nsize; ++n){
			for(size_t i=1UL; i<=8; ++i){
				std::stringstream path;
				path << "data/" << names[n] << "/" << names[n] << "." << (i * 2) << ".jpg";
				files.push_back(path.str());
			}
		}
	}

	std::vector<std::string> filtersFiles;
	filtersFiles.push_back("data/test/none.png");
	filtersFiles.push_back("data/test/blur1.png");
	filtersFiles.push_back("data/test/sharp1.png");

	bool color = true;

	// Load images
	std::shared_ptr<cnn::ImageBatch<uchar>> b = cnn::ImageBatch<uchar>::fromFiles(files, color);
	std::shared_ptr<cnn::ImageBatch<uchar>> filtersUchar = cnn::ImageBatch<uchar>::fromFiles(filtersFiles, color);

	hipDeviceProp_t prop0;
	hipGetDeviceProperties(&prop0, 0);

	hipSetDevice(cnn::config::Cuda::CUDA_DEVICE_ID);

	bool dof = false;
	if (dof){
		doFloat(b, filtersUchar);
	} else {
		doUchar(b, filtersUchar);
	}

    return 0;
}



void doUchar(
	std::shared_ptr<cnn::ImageBatch<uchar>>& pImages, 
	std::shared_ptr<cnn::ImageBatch<uchar>>& pKernels)
{
	{
		// space for uchars and floats
		cnn::gpu::GpuBuffer bImages;
		bImages.allocate(pImages->getBatchByteSize());
		bImages.writeToDevice(pImages->getBatchDataPtr(), pImages->getBatchByteSize());
		assert(hipDeviceSynchronize() == hipSuccess);

		// centering
		cnn::gpu::GpuBuffer bCenterImage;
		bCenterImage.allocate(pImages->getAlignedImageByteSize() * sizeof(float));

		cnn::gpu::AverageCenterizer<uchar> cent;
		cent.build(*pImages, bImages, bCenterImage);
		assert(hipDeviceSynchronize() == hipSuccess);

		cnn::gpu::GpuBuffer bCenterized;
		bCenterized.allocate(bImages.getByteSize());
		assert(hipDeviceSynchronize() == hipSuccess);

		cent.normalize(*pImages, bImages, bCenterImage, bCenterized);
		assert(hipDeviceSynchronize() == hipSuccess);

		// sharp
		cnn::gpu::Sharpener<uchar> shrp;
		shrp.build(*pImages, bCenterized, bCenterImage);
		assert(hipDeviceSynchronize() == hipSuccess);

		shrp.normalize(*pImages, bCenterized, bCenterImage, bCenterized);
		assert(hipDeviceSynchronize() == hipSuccess);

		// convolution
		cnn::gpu::GpuBuffer bKernels;
		bKernels.allocate(pKernels->getBatchByteSize());
		bKernels.writeToDevice(pKernels->getBatchDataPtr(), pKernels->getBatchByteSize());

		cnn::ImageBatch<uchar> filtered(178, 198, pImages->getImageChannelsCount());
		filtered.allocateSpaceForImages(pImages->getImagesCount() * pKernels->getImagesCount(), true);

		cnn::gpu::GpuBuffer bFilteredBuffer;
		bFilteredBuffer.allocate(filtered.getBatchByteSize());
		assert(hipDeviceSynchronize() == hipSuccess);

		cnn::gpu::ImageConvolution<uchar> sc;
		sc.compute(*pImages, bCenterized, *pKernels, bKernels, filtered, bFilteredBuffer, 1, 1);
		assert(hipDeviceSynchronize() == hipSuccess);

		// load
		cnn::ImageBatch<uchar> centerized(pImages->getImageWidth(), pImages->getImageHeight(), pImages->getImageChannelsCount());
		centerized.allocateSpaceForImages(pImages->getImagesCount(), true);
		bCenterized.loadFromDevice(centerized.getBatchDataPtr(), centerized.getBatchByteSize());
		bFilteredBuffer.loadFromDevice(filtered.getBatchDataPtr(), filtered.getBatchByteSize());
		assert(hipDeviceSynchronize() == hipSuccess);

		// show
		cv::namedWindow("raw images");
		cv::namedWindow("centered images");
		cv::namedWindow("kerneled images");
		cv::namedWindow("kernels");

		cv::imshow("raw images", pImages->retriveAllImagesAsMat(5));
		cv::imshow("centered images", centerized.retriveAllImagesAsMat(5));
		cv::imshow("kerneled images", filtered.retriveAllImagesAsMat(pKernels->getImagesCount()));
		cv::imshow("kernels", pKernels->retriveAllImagesAsMat(pKernels->getImagesCount()));
	}
	cv::waitKey(0);
}


void doFloat(
	std::shared_ptr<cnn::ImageBatch<uchar>>& b, 
	std::shared_ptr<cnn::ImageBatch<uchar>>& filtersUchar)
{
	cnn::ImageBatch<float> fb(b->getImageWidth(), b->getImageHeight(), b->getImageChannelsCount(), 32 * sizeof(float));
	fb.allocateSpaceForImages(b->getImagesCount(), true);

	{
		// space for uchars and floats
		cnn::gpu::GpuBuffer uchars;
		uchars.allocate(b->getBatchByteSize());
		uchars.writeToDevice(b->getBatchDataPtr(), b->getBatchByteSize());

		cnn::gpu::GpuBuffer floats;
		floats.allocate(b->getBatchByteSize() * sizeof(float));
		assert(hipDeviceSynchronize() == hipSuccess);

		// convert
		cnn::gpu::Converter<uchar, float> converter;
		converter.convert(*b, uchars, floats);
		assert(hipDeviceSynchronize() == hipSuccess); 

		// convert filters
		uchars.writeToDevice(filtersUchar->getBatchDataPtr(), filtersUchar->getBatchByteSize());
		
		cnn::gpu::GpuBuffer kernels;
		kernels.allocate(filtersUchar->getBatchByteSize() * sizeof(float));
		converter.convert(*filtersUchar, uchars, kernels);
		assert(hipDeviceSynchronize() == hipSuccess);  

		// centering
		cnn::gpu::GpuBuffer centerImage;
		centerImage.allocate(b->getAlignedImageByteSize() * sizeof(float));

		cnn::gpu::AverageCenterizer<float> cent;
		cent.build(fb, floats, centerImage);
		assert(hipDeviceSynchronize() == hipSuccess);

		cnn::gpu::GpuBuffer centerized;
		centerized.allocate(floats.getByteSize());
		assert(hipDeviceSynchronize() == hipSuccess);

		cent.normalize(fb, floats, centerImage, centerized);
		assert(hipDeviceSynchronize() == hipSuccess);

		// sharp
		cnn::gpu::Sharpener<float> shrp;
		shrp.build(fb, centerized, centerImage);
		assert(hipDeviceSynchronize() == hipSuccess);

		shrp.normalize(fb, centerized, centerImage, centerized);
		assert(hipDeviceSynchronize() == hipSuccess);

		// convolution
		cnn::ImageBatch<float> filtered(35, 39, b->getImageChannelsCount(), 32 * sizeof(float));
		filtered.allocateSpaceForImages(b->getImagesCount() * filtersUchar->getImagesCount(), true);

		cnn::gpu::GpuBuffer filteredBuffer;
		filteredBuffer.allocate(filtered.getBatchByteSize());
		assert(hipDeviceSynchronize() == hipSuccess);

		
		cnn::ImageBatch<float> filters(filtersUchar->getImageWidth(), filtersUchar->getImageHeight(), filtersUchar->getImageChannelsCount(), 32 * sizeof(float));
		filters.allocateSpaceForImages(filtersUchar->getImagesCount(), true);

		cnn::gpu::ImageConvolution<float> sc;
		sc.compute(fb, centerized, filters, kernels, filtered, filteredBuffer, 5, 5);
		assert(hipDeviceSynchronize() == hipSuccess);

		// unconvert centerized
		cnn::gpu::Converter<float, uchar> converter2;
		converter2.convert(fb, centerized, uchars);
		assert(hipDeviceSynchronize() == hipSuccess);

		uchars.loadFromDevice(b->getBatchDataPtr(), b->getBatchByteSize());
		assert(hipDeviceSynchronize() == hipSuccess);

		// unconvert filtered
		converter2.convert(filtered, filteredBuffer, uchars);
		assert(hipDeviceSynchronize() == hipSuccess);

		cnn::ImageBatch<uchar> filterResult(filtered.getImageWidth(), filtered.getImageHeight(), filtered.getImageChannelsCount());
		filterResult.allocateSpaceForImages(b->getImagesCount() * filters.getImagesCount(), true);
		uchars.loadFromDevice(filterResult.getBatchDataPtr(), filterResult.getBatchByteSize());
		assert(hipDeviceSynchronize() == hipSuccess);

		// show
		cv::namedWindow("centered images");
		cv::namedWindow("kerneled images");
		cv::namedWindow("kernels");

		cv::imshow("centered images", b->retriveAllImagesAsMat(5));
		cv::imshow("kerneled images", filterResult.retriveAllImagesAsMat(filters.getImagesCount()));
		cv::imshow("kernels", filtersUchar->retriveAllImagesAsMat(filters.getImagesCount()));
	}
	cv::waitKey(0);
}