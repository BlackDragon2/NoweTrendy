
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <fstream>
#include <iostream>

#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>

#include "ImagesBatch.h"
#include "Kernels.h"


int main()
{
	std::vector<std::string> files;
	files.push_back("data/test2.jpg");
	files.push_back("data/test3.jpg");

	cnn::ImagesBatch b = cnn::ImagesBatch::fromFiles(files);
	{
		cv::namedWindow("some name", CV_WINDOW_AUTOSIZE);
		cv::namedWindow("some name2", CV_WINDOW_AUTOSIZE);
		cv::namedWindow("some name23", CV_WINDOW_AUTOSIZE);
		cv::imshow("some name", b.getImageAsMat(0));
		cv::imshow("some name2", b.getImageAsMat(1));

		// <<<216, 500>>>
		// 216 blocks per 500 threads
		uchar* imgsOnDev;
		hipMalloc<uchar>(&imgsOnDev, b.getBatchSize());
		hipMemcpy(imgsOnDev, b.getImagesData(), b.getBatchSize(), hipMemcpyHostToDevice);
		centerImages<<<216, 500>>>(imgsOnDev, b.getImageSize(), b.getImagesCount());
		hipMemcpy(b.getImagesData(), imgsOnDev, b.getBatchSize(), hipMemcpyDeviceToHost);
		hipFree(imgsOnDev);

		cv::imshow("some name23", b.getImageAsMat(1));
	}
	cv::waitKey(0);

    return 0;
}