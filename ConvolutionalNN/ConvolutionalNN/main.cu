
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <fstream>
#include <iostream>
#include <sstream>
#include <Windows.h>

#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>

#include "ImagesBatch.h"
#include "GPU/Normalizations.cuh"
#include "GPU/GpuBuffer.cuh"
#include "Types.h"
#include "Utils/FoldsFactory.h"


int main()
{
	std::shared_ptr<std::vector<size_t>> folds = cnn::utils::FoldsFactory::prepareFoldVector(117, 7, cnn::utils::FoldsFactory::FitTactic::DEFAULT);
	std::shared_ptr<std::vector<size_t>> folds2 = cnn::utils::FoldsFactory::prepareFoldVector(117, 7, cnn::utils::FoldsFactory::FitTactic::CUT);
	std::shared_ptr<std::vector<size_t>> folds3 = cnn::utils::FoldsFactory::prepareFoldVector(117, 7, cnn::utils::FoldsFactory::FitTactic::EXTEND);
	std::shared_ptr<std::vector<size_t>> folds4 = cnn::utils::FoldsFactory::prepareFoldVector(117, 7, cnn::utils::FoldsFactory::FitTactic::EXTEND_WITH_COPIES);

	std::vector<std::string> files;
	//for(size_t a=0UL; a<10UL; ++a){
		for(size_t i=1UL; i<=20; ++i){
			std::stringstream path;
			path << "data/phughe/phughe." << i << ".jpg";
			files.push_back(path.str());
		}
	//}

	__int64 freq, s1, s12, s2, e1, e2;
	QueryPerformanceFrequency(reinterpret_cast<LARGE_INTEGER*>(&freq));
	double spc = 1.0 / freq;

	std::shared_ptr<cnn::ImagesBatch<uint>> b = cnn::ImagesBatch<uint>::fromFiles(files);
	{
		cv::namedWindow("some name23", CV_WINDOW_AUTOSIZE);

		QueryPerformanceCounter(reinterpret_cast<LARGE_INTEGER*>(&s1));

		cnn::gpu::GpuBuffer<uint> devbuffer(b->getBatchUnitSize());

		QueryPerformanceCounter(reinterpret_cast<LARGE_INTEGER*>(&s12));
		
		devbuffer.writeToDevice(b->getImagesData(), b->getBatchUnitSize());
		assert(hipDeviceSynchronize() == hipSuccess);
		
		QueryPerformanceCounter(reinterpret_cast<LARGE_INTEGER*>(&s2));
		
		cnn::gpu::Normalizations<uint>::centerize(b, devbuffer);
		
		assert(hipDeviceSynchronize() == hipSuccess);
		
		QueryPerformanceCounter(reinterpret_cast<LARGE_INTEGER*>(&e1));
		
		devbuffer.loadFromDevice(b->getImagesData(), b->getBatchUnitSize());

		QueryPerformanceCounter(reinterpret_cast<LARGE_INTEGER*>(&e2));
		
		std::cout << "allocation:     " << double(s12 - s1) * spc << std::endl;
		std::cout << "send:           " << double(s2 - s12) * spc << std::endl;
		std::cout << "comp:           " << double(e1 - s2) * spc << std::endl;
		std::cout << "recv & dealloc: " << double(e2 - e1) * spc << std::endl;
		std::cout << "all:            " << double(e2 - s1) * spc << std::endl;

		cv::imshow("some name23", b->getImageAsMat(17));
	}
	cv::waitKey(0);

    return 0;
}