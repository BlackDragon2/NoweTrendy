
#include "GpuBuffer.cuh"

#include "../Utils/Utils.h"


namespace cnn {
	namespace gpu {


GpuBuffer::GpuBuffer()
:
	mAddress(nullptr),
	mByteSize(0UL),
	mByteAlignment(0UL)
{

}


GpuBuffer::GpuBuffer(size_t pBytesCount, size_t pByteAlignment)
:
	mAddress(nullptr),
	mByteSize(0UL),
	mByteAlignment(0UL)
{
	allocate(pBytesCount, pByteAlignment);
}


GpuBuffer::~GpuBuffer(){
	free();
}


void GpuBuffer::allocate(size_t pBytesCount, size_t pByteAlignment){
	mByteAlignment	= pByteAlignment;
	mByteSize		= utils::align(pBytesCount, pByteAlignment);
	hipError_t result = hipMalloc<uchar>(&mAddress, mByteSize);
	assert(result == hipSuccess);
}


void GpuBuffer::free(){
	if(mAddress != nullptr){
		hipError_t result = hipFree(reinterpret_cast<void*>(mAddress));
		assert(result == hipSuccess);
		mAddress		= nullptr;
		mByteSize		= 0UL;
		mByteAlignment	= 0UL;
	}
}


void GpuBuffer::reallocate(size_t pBytesCount, size_t pByteAlignment){
	free();
	allocate(pBytesCount, pByteAlignment);
}


size_t GpuBuffer::getByteSize() const {
	return mByteSize;
}


size_t GpuBuffer::getAlignment() const {
	return mByteAlignment;
}


	}
}